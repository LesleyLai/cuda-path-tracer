#include "hip/hip_runtime.h"
#include "path_tracer.hpp"

#include "camera.hpp"
#include "cuda_buffer.hpp"
#include "distributions.cuh"
#include "span.hpp"

#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <thrust/random.h>

#include <cmath>
#include <fmt/format.h>

#include <iterator>

#include <assimp/Importer.hpp>
#include <assimp/postprocess.h>
#include <assimp/scene.h>

#include <glm/gtx/compatibility.hpp>

#include "intersections.cuh"

static constexpr Material mat[] = {{Material::Type::Diffuse, 0},
                                   {Material::Type::Diffuse, 1},
                                   {Material::Type::Dielectric, 0},
                                   {Material::Type::Metal, 0}};

static const DiffuseMateral diffuse_mat[] = {{{0.8, 0.8, 0.0}},
                                             {{0.1, 0.2, 0.5}}};
static const MetalMaterial metal_mat[] = {{{0.8, 0.6, 0.2}, 1.0}};
static const DielectricMaterial dielectric_mat[] = {{1.5}};

void check_CUDA_error(std::string_view msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fmt::print(stderr, "Cuda error: {}: {}.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

struct Index2D {
  unsigned int x = 0;
  unsigned int y = 0;
};

[[nodiscard]] __device__ constexpr auto calculate_index_2d() -> Index2D
{
  const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
  return Index2D{x, y};
}

[[nodiscard]] __device__ auto raygen(glm::mat4 camera_matrix, float fov,
                                     unsigned int width, unsigned int height,
                                     unsigned int x, unsigned int y,
                                     thrust::default_random_engine& rng) -> Ray
{
  const float aspect_ratio =
      static_cast<float>(width) / static_cast<float>(height);

  const float viewport_height = 2.0f * tan(fov / 2);
  const float viewport_width = aspect_ratio * viewport_height;
  const float focal_length = 1.0;

  const auto origin = glm::vec3(0, 0, 0);
  const auto horizontal = glm::vec3(viewport_width, 0, 0);
  const auto vertical = glm::vec3(0, viewport_height, 0);
  const auto lower_left_corner = origin - horizontal / 2.f - vertical / 2.f -
                                 glm::vec3(0, 0, focal_length);

  thrust::uniform_real_distribution<float> dist(0.0, 1.0);

  const auto u =
      (static_cast<float>(x) + dist(rng)) / static_cast<float>(width - 1);
  const auto v =
      (static_cast<float>(y) + dist(rng)) / static_cast<float>(height - 1);
  const auto direction =
      lower_left_corner + u * horizontal + v * vertical - origin;

  const auto world_origin = glm::vec3(camera_matrix * glm::vec4(origin, 1.0));
  const auto world_direction =
      glm::normalize(glm::vec3(camera_matrix * glm::vec4(direction, 0.0)));
  return Ray{world_origin, 1e-4, world_direction, FLT_MAX};
}

__device__ auto get_background_color(Ray r) -> glm::vec3
{
  const glm::vec3 unit_direction = glm::normalize(r.direction);
  const auto t = 0.5f * (unit_direction.y + 1.0f);
  return glm::lerp(glm::vec3(0.5, 0.7, 1.0), glm::vec3(1.0, 1.0, 1.0), t);
}

__device__ auto ray_mesh_intersection_test(Ray ray, const Vertex* vertices,
                                           Span<const std::uint32_t> indices,
                                           HitRecord& record) -> bool
{
  bool hit = false;
  for (std::size_t j = 0; j < indices.size(); j += 3) {
    const auto index0 = indices[j];
    const auto index1 = indices[j + 1];
    const auto index2 = indices[j + 2];

    const auto p0 = vertices[index0].position;
    const auto p1 = vertices[index1].position;
    const auto p2 = vertices[index2].position;

    if (ray_triangle_intersection_test(ray, p0, p1, p2, record)) {
      hit = true;
      ray.t_max = record.t;
    }
  }
  return hit;
}

__device__ auto ray_object_intersection_test(Ray ray, GPUObject obj,
                                             AggregateView aggregate,
                                             const Vertex* vertices,
                                             Span<const std::uint32_t> indices,
                                             HitRecord& record) -> bool
{
  switch (obj.type) {
  case ObjectType::sphere: {
    const auto sphere = aggregate.spheres[obj.index];
    return ray_sphere_intersection_test(ray, sphere, record);
  }
  case ObjectType::triangle: {
    const auto triangle = aggregate.triangles[obj.index];
    return ray_triangle_intersection_test(ray, triangle.pt0, triangle.pt1,
                                          triangle.pt2, record);
  }
  case ObjectType::mesh:
    return ray_mesh_intersection_test(ray, vertices, indices, record);
  }
  // unreachable
  return false;
}

__device__ auto ray_scene_intersection_test(Ray ray, AggregateView aggregate,
                                            const Vertex* vertices,
                                            Span<const std::uint32_t> indices,
                                            HitRecord& record) -> bool
{
  bool hit = false;

  const auto objects = aggregate.objects;
  const auto* object_material_indices = aggregate.object_material_indices;

  for (std::size_t i = 0; i < objects.size(); ++i) {
    const GPUObject obj = objects[i];
    if (ray_object_intersection_test(ray, obj, aggregate, vertices, indices,
                                     record)) {
      hit = true;
      record.material_id = object_material_indices[i];
      ray.t_max = record.t;
    }
  }

  return hit;
}

[[nodiscard]] __host__ __device__ constexpr auto hash(unsigned int a)
    -> unsigned int
{
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

__device__ static auto reflectance(float cosine, float ref_idx) -> float
{
  // Use Schlick's approximation for reflectance.
  auto r0 = (1 - ref_idx) / (1 + ref_idx);
  r0 = r0 * r0;
  return r0 + (1 - r0) * pow((1 - cosine), 5);
}

__device__ void evaluate_material(Ray& ray, const HitRecord record,
                                  thrust::default_random_engine& rng,
                                  glm::vec3& color, Span<const Material> mat,
                                  Span<const DiffuseMateral> diffuse_mat,
                                  Span<const MetalMaterial> metal_mat,
                                  Span<const DielectricMaterial> dielectric_mat)
{
  ray.origin = record.point - 1e-4f *
                                  glm::sign(dot(ray.direction, record.normal)) *
                                  record.normal;
  // material stuff
  const Material& material = mat[record.material_id];
  switch (material.type) {
  case Material::Type::Diffuse: {
    auto scatter_direction =
        glm::normalize(record.normal + random_in_unit_sphere(rng));

    // Catch degenerated case
    if (abs(scatter_direction.x) < 1e-8 && abs(scatter_direction.y) < 1e-8 &&
        abs(scatter_direction.z) < 1e-8) {
      scatter_direction = record.normal;
    }

    ray.direction = scatter_direction;
    color *= diffuse_mat[material.index].albedo;
  } break;
  case Material::Type::Metal: {
    const auto metal = metal_mat[material.index];
    const auto reflected = glm::reflect(ray.direction, record.normal);
    const auto scatter_direction =
        reflected + metal.fuzz * random_in_unit_sphere(rng);
    ray.direction = scatter_direction;
    if (dot(scatter_direction, record.normal) > 0) {
      color *= metal.albedo;
    } else {
      color = glm::vec3(0.0, 0.0, 0.0);
    }
  } break;
  case Material::Type::Dielectric: {
    const auto dielectric = dielectric_mat[material.index];
    const auto refraction_ratio = record.side == HitFaceSide::front
                                      ? (1.0f / dielectric.refraction_index)
                                      : dielectric.refraction_index;

    const auto unit_direction = normalize(ray.direction);
    const float cos_theta = min(dot(-unit_direction, record.normal), 1.0f);
    const float sin_theta = sqrtf(1.0f - cos_theta * cos_theta);

    const bool cannot_refract = refraction_ratio * sin_theta > 1.0;
    thrust::uniform_real_distribution<float> dist(0.0, 1.0);
    const glm::vec3 direction = [&]() {
      if (cannot_refract ||
          reflectance(cos_theta, refraction_ratio) > dist(rng)) {
        return reflect(unit_direction, record.normal);
      } else {
        return refract(unit_direction, record.normal, refraction_ratio);
      }
    }();

    ray = Ray{record.point, 1e-5, direction, std::numeric_limits<float>::max()};
  } break;
  }
}

[[nodiscard]] __device__ auto gamma_correction(glm::vec3 color) -> glm::vec3
{
  color.x = glm::pow(color.x, 1.f / 2.2f);
  color.y = glm::pow(color.y, 1.f / 2.2f);
  color.z = glm::pow(color.z, 1.f / 2.2f);
  return color;
}

__global__ void path_tracing_kernel(
    unsigned int width, unsigned int height, glm::mat4 camera_matrix, float fov,
    glm::vec3* color_buffer, glm::vec3* normal_buffer,
    glm::vec3* position_buffer, std::size_t iteration, AggregateView aggregate,
    Span<const Material> mat, Span<const DiffuseMateral> diffuse_mat,
    Span<const MetalMaterial> metal_mat,
    Span<const DielectricMaterial> dielectric_mat, const Vertex* vertices,
    Span<const std::uint32_t> indices)
{
  const auto [x, y] = calculate_index_2d();
  if (x >= width || y >= height) return;
  const auto index = x + ((height - y) * width);

  thrust::default_random_engine rng(hash(hash(index) ^ iteration));

  auto ray = raygen(camera_matrix, fov, width, height, x, y, rng);

  // Path tracing
  glm::vec3 color{1.0f, 1.0f, 1.0f};
  glm::vec3 normal{0.f, 0.f, 0.f};
  glm::vec3 position{INFINITY, INFINITY, INFINITY};

  for (int i = 0; i < 50; ++i) {
    HitRecord record;
    const bool hit =
        ray_scene_intersection_test(ray, aggregate, vertices, indices, record);
    if (!hit) {
      color *= get_background_color(ray);
      break;
    }

    evaluate_material(ray, record, rng, color, mat, diffuse_mat, metal_mat,
                      dielectric_mat);
    if (i == 0) {
      normal = record.normal;
      position = record.point;
    }
  }

  color = gamma_correction(color);

  // Final gathering
  const auto sample_count = static_cast<float>(iteration + 1);
  color_buffer[index] =
      (color_buffer[index] * (sample_count - 1) + color) / sample_count;
  normal_buffer[index] =
      (normal_buffer[index] * (sample_count - 1) + normal) / sample_count;
  position_buffer[index] = position;
}

enum class BufferNormalizationMethod { none, neg1_1_to_0_1 };

__global__ void preview_kernel(unsigned int width, unsigned int height,
                               BufferNormalizationMethod normalization_method,
                               glm::vec3* buffer, uchar4* pbo)
{
  const auto [x, y] = calculate_index_2d();
  if (x >= width || y >= height) return;
  const auto index = x + ((height - y) * width);

  auto color = buffer[index];

  switch (normalization_method) {
  case BufferNormalizationMethod::neg1_1_to_0_1: color = color * 0.5f + 0.5f;
  default: break;
  }

  constexpr auto color_float_to_255 = [](float v) {
    return static_cast<unsigned char>(glm::clamp(v, 0.f, 1.f) * 255.99f);
  };

  if (x <= width && y <= height) {
    pbo[index] =
        uchar4{color_float_to_255(color.x), color_float_to_255(color.y),
               color_float_to_255(color.z), 1};
  }
}

[[nodiscard]] static auto load_obj(const char* filename) -> GPUMesh
{
  Assimp::Importer importer;

  const aiScene* scene = importer.ReadFile(filename, aiProcess_Triangulate);
  if (!scene || !scene->HasMeshes()) {
    throw std::runtime_error(fmt::format("Unable to load {}", filename));
  }
  const aiMesh* mesh = scene->mMeshes[0];

  thrust::host_vector<Vertex> vertices;
  for (unsigned i = 0; i != mesh->mNumVertices; i++) {
    const aiVector3D v = mesh->mVertices[i];
    // const aiVector3D n = mesh->mNormals[i];
    // const aiVector3D t = mesh->mTextureCoords[0][i];
    vertices.push_back(Vertex{{v.x + 10.f, v.z, v.y}});
  }

  thrust::host_vector<std::uint32_t> indices;
  for (unsigned i = 0; i != mesh->mNumFaces; i++)
    for (unsigned j = 0; j != 3; j++)
      indices.push_back(mesh->mFaces[i].mIndices[j]);

  GPUMesh mesh_gpu;
  mesh_gpu.vertices = cuda::make_buffer<Vertex>(vertices.size());
  mesh_gpu.indices = cuda::make_buffer<std::uint32_t>(indices.size());
  mesh_gpu.indices_count = indices.size();

  thrust::copy(vertices.begin(), vertices.end(),
               thrust::device_pointer_cast(mesh_gpu.vertices.data()));
  thrust::copy(indices.begin(), indices.end(),
               thrust::device_pointer_cast(mesh_gpu.indices.data()));

  return mesh_gpu;
}

PathTracer::PathTracer()
{
  cube_ = load_obj("models/cube.obj");
}

void PathTracer::path_trace(uchar4* dev_pbo, const Camera& camera,
                            unsigned int width, unsigned int height)
{
  if (iteration_ >= max_iterations) return;

  constexpr unsigned int block_size = 16;
  const dim3 threads_per_block(block_size, block_size);

  const auto blocks_x = (width + block_size - 1) / block_size;
  const auto blocks_y = (height + block_size - 1) / block_size;
  const dim3 full_blocks_per_grid(blocks_x, blocks_y);

  path_tracing_kernel<<<full_blocks_per_grid, threads_per_block>>>(
      width, height, camera.camera_matrix(), camera.fov(),
      dev_color_buffer_.data(), dev_normal_buffer_.data(),
      dev_position_buffer_.data(), iteration_, AggregateView{aggregate_},
      Span{dev_mat_.data(), std::size(mat)},
      Span{dev_diffuse_mat_.data(), std::size(diffuse_mat)},
      Span{dev_metal_mat_.data(), std::size(metal_mat)},
      Span{dev_dielectric_mat_.data(), std::size(dielectric_mat)},
      cube_.vertices.data(), Span{cube_.indices.data(), cube_.indices_count});
  check_CUDA_error("Path Tracing kernel");

  switch (display_buffer_) {
  case DisplayBuffer::path_tracing:
    preview_kernel<<<full_blocks_per_grid, threads_per_block>>>(
        width, height, BufferNormalizationMethod::none,
        dev_color_buffer_.data(), dev_pbo);
    break;
  case DisplayBuffer::normal:
    preview_kernel<<<full_blocks_per_grid, threads_per_block>>>(
        width, height, BufferNormalizationMethod::neg1_1_to_0_1,
        dev_normal_buffer_.data(), dev_pbo);
    break;
  case DisplayBuffer::position:
    preview_kernel<<<full_blocks_per_grid, threads_per_block>>>(
        width, height, BufferNormalizationMethod::none,
        dev_position_buffer_.data(), dev_pbo);
    break;
  }
  check_CUDA_error("Preview kernel");

  CUDA_CHECK(hipDeviceSynchronize());

  ++iteration_;
}

void PathTracer::restart()
{
  iteration_ = 0;
}

void PathTracer::resize_image(unsigned int width, unsigned int height)
{
  const auto image_size = width * height;
  dev_color_buffer_ = cuda::make_buffer<glm::vec3>(image_size);
  dev_normal_buffer_ = cuda::make_buffer<glm::vec3>(image_size);
  dev_position_buffer_ = cuda::make_buffer<glm::vec3>(image_size);
  CUDA_CHECK(hipDeviceSynchronize());
  restart();
}

void PathTracer::create_buffers(unsigned int width, unsigned int height,
                                const SceneBuilder& scene)
{
  aggregate_ = scene.build();
  dev_mat_ = cuda::create_buffer_from_cpu_data(Span{mat});
  dev_diffuse_mat_ = cuda::create_buffer_from_cpu_data(Span{diffuse_mat});
  dev_metal_mat_ = cuda::create_buffer_from_cpu_data(Span{metal_mat});
  dev_dielectric_mat_ = cuda::create_buffer_from_cpu_data(Span{dielectric_mat});
  resize_image(width, height);
}
