#include "hip/hip_runtime.h"
#include "path_tracer.hpp"
#include <cstddef>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

#include <cmath>
#include <fmt/format.h>

#include <glm/gtx/compatibility.hpp>

static const Sphere spheres[] = {{{0.0f, 0.0f, -1.0f}, 0.5f},
                                 {{0.0f, -100.5f, -1.0f}, 100.f}};

void check_CUDA_error(std::string_view msg)
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fmt::print(stderr, "Cuda error: {}: {}.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

struct Index2D {
  unsigned int x = 0;
  unsigned int y = 0;
};

[[nodiscard]] __device__ auto calculate_index_2d()
{
  const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
  const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
  return Index2D{x, y};
}

[[nodiscard]] __device__ auto raygen(unsigned int width, unsigned int height,
                                     unsigned int x, unsigned int y) -> Ray
{
  const float aspect_ratio =
      static_cast<float>(width) / static_cast<float>(height);

  const float viewport_height = 2.0f;
  const float viewport_width = aspect_ratio * viewport_height;
  const float focal_length = 1.0;

  const auto origin = glm::vec3(0, 0, 0);
  const auto horizontal = glm::vec3(viewport_width, 0, 0);
  const auto vertical = glm::vec3(0, viewport_height, 0);
  const auto lower_left_corner = origin - horizontal / 2.f - vertical / 2.f -
                                 glm::vec3(0, 0, focal_length);

  const auto u = static_cast<float>(x) / static_cast<float>(width - 1);
  const auto v = static_cast<float>(y) / static_cast<float>(height - 1);
  return Ray{origin,
             lower_left_corner + u * horizontal + v * vertical - origin};
}

__device__ auto get_background_color(Ray r) -> glm::vec3
{
  const glm::vec3 unit_direction = glm::normalize(r.direction);
  const auto t = 0.5f * (unit_direction.y + 1.0f);
  return glm::lerp(glm::vec3(0.5, 0.7, 1.0), glm::vec3(1.0, 1.0, 1.0), t);
}

__global__ void path_tracing_kernel(uchar4* pbo, Sphere* spheres,
                                    std::size_t sphere_count,
                                    unsigned int width, unsigned int height)
{
  const auto [x, y] = calculate_index_2d();
  if (x >= width || y >= height) return;
  const auto index = x + ((height - y) * width);

  const auto ray = raygen(width, height, x, y);

  HitRecord record;
  bool hit = false;
  float t_max = std::numeric_limits<float>::max();
  for (std::size_t i = 0; i < sphere_count; ++i) {
    const auto& sphere = spheres[i];
    HitRecord new_record;
    if (ray_sphere_intersection_test(ray, sphere.center, sphere.radius,
                                     new_record)) {
      hit = true;
      if (new_record.t < t_max) {
        record = new_record;
        t_max = new_record.t;
      }
    }
  }

  const glm::vec3 color =
      hit ? ((record.normal + 1.0f) * 0.5f) : get_background_color(ray);

  constexpr auto normalize_color = [](float v) {
    return static_cast<unsigned char>(v * 255.99f);
  };

  if (x <= width && y <= height) {
    pbo[index].w = 1;
    pbo[index].x = normalize_color(color.x);
    pbo[index].y = normalize_color(color.y);
    pbo[index].z = normalize_color(color.z);
  }
}

PathTracer::PathTracer() = default;

void PathTracer::path_trace(uchar4* PBOpos, unsigned int width,
                            unsigned int height)
{
  constexpr unsigned int block_size = 16;
  const dim3 threads_per_block(block_size, block_size);

  const auto blocks_x = (width + block_size - 1) / block_size;
  const auto blocks_y = (height + block_size - 1) / block_size;
  const dim3 full_blocks_per_grid(blocks_x, blocks_y);

  path_tracing_kernel<<<full_blocks_per_grid, threads_per_block>>>(
      PBOpos, dev_spheres_.data(), std::size(spheres), width, height);
  check_CUDA_error("Visualization kernel");

  CUDA_CHECK(hipDeviceSynchronize());
}

void PathTracer::create_buffers()
{
  dev_spheres_ = cuda::make_buffer<Sphere>(std::size(spheres));
  CUDA_CHECK(hipMemcpy(dev_spheres_.data(), spheres,
                        std::size(spheres) * sizeof(Sphere),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
}